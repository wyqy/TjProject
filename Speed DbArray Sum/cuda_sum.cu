#include "hip/hip_runtime.h"
#include "cuda_sum.cuh"

__host__ hipError_t initialCuda(int device)
{
    // ��ʼ��CUDA�豸, �̼߳���!
    hipError_t cudaStatus;

    // �����������
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n[Error] last execution failed: %s!\n", hipGetErrorString(cudaStatus));
    }

    // ȷ��CUDA�豸, Ĭ��ֻѡ�е�һ���豸
    cudaStatus = hipSetDevice(device);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n[Error] hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
    }

    return cudaStatus;
}


__host__ hipError_t sumWithCuda(float* retValue, size_t* retLen, const float* data, size_t len)
{
    float* lastResult = nullptr;
    float* nowDataA = nullptr;
    float* nowDataB = nullptr;
    float* nowResult = nullptr;
    bool   isLastResultAMalloc = false;
    bool   isNowResultMalloc = false;

    bool isOdd = false;
    size_t wholeDataLen = len;
    size_t opDataLen = 0;
    size_t resDataLen = 0;
    size_t resMemLen = 0;

    dim3 dimBlock(BLOCK_DIMONE_x1024);
    dim3 dimGrid(1);

    hipError_t cudaStatus;

    // �����������
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n[Error] last execution failed: %s!\n", hipGetErrorString(cudaStatus));
        goto Error;
    }


    // ���������Դ�
    // ��������size
    resDataLen = wholeDataLen;
    // ����grid��blocks��size
    dimGrid.x = ((unsigned int)resDataLen % BLOCK_DIMONE_x1024 != 0) ? ((unsigned int)resDataLen / BLOCK_DIMONE_x1024 + 1) : ((unsigned int)resDataLen / BLOCK_DIMONE_x1024);
    if ((unsigned int)resDataLen < BLOCK_DIMONE_x1024) dimBlock.x = (unsigned int)resDataLen;
    else dimBlock.x = BLOCK_DIMONE_x1024;
    // �����ڴ�size
    resMemLen = max((size_t)(dimGrid.x * dimBlock.x), resDataLen);


    // �����������: host -> device
    // ��ʼ��lastResult�ڴ�
    cudaStatus = hipMalloc((void**)&lastResult, resMemLen * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n[Error] hipMalloc failed for source!\n");
        goto Error;
    }
    else isLastResultAMalloc = true;
    // ����lastResult�ڴ�
    cudaStatus = hipMemcpy(lastResult, data, wholeDataLen * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n[Error] hipMemcpy failed when copy dataA for the first time!\n");
        goto Error;
    }
    // ����������
    nowDataA = lastResult;
    nowDataB = lastResult + opDataLen;
    // ��ʼ��nowResult�ڴ�
    cudaStatus = hipMalloc((void**)&nowResult, resMemLen * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n[Error] hipMalloc failed for result!\n");
        goto Error;
    }
    else isNowResultMalloc = true;


    // ����read kernel
    readKernel <<<dimGrid, dimBlock>>> (nowResult, lastResult);
    // ���kernelִ�д���
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n[Error] kernel launching failed: %s @ %d, %d!\n", hipGetErrorString(cudaStatus), dimGrid.x, dimBlock.x);
        goto Error;
    }
    // �ȴ�kernelִ�����
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n[Error] hipDeviceSynchronize returned error code %d after launching kernel!\n", cudaStatus);
        goto Error;
    }


    // ��ʼ���
    // �鲢��: ������С!
    while (wholeDataLen > CUDA_MAX_RESLEN)
    {
        // ��������size
        isOdd = (wholeDataLen % 2 != 0);
        opDataLen = wholeDataLen / 2;
        resDataLen = isOdd ? opDataLen + 1 : opDataLen;
        // ����grid��blocks��size
        dimGrid.x = ((unsigned int)opDataLen % BLOCK_DIMONE_x1024 != 0) ? ((unsigned int)opDataLen / BLOCK_DIMONE_x1024 + 1) : ((unsigned int)opDataLen / BLOCK_DIMONE_x1024);
        if ((unsigned int)opDataLen < BLOCK_DIMONE_x1024) dimBlock.x = (unsigned int)opDataLen;
        else dimBlock.x = BLOCK_DIMONE_x1024;
        // �����ڴ�size
        resMemLen = max((size_t)(dimGrid.x * dimBlock.x), resDataLen);

        // �������ڴ洦��
        // �ͷ�lastResult�ڴ�
        cudaStatus = hipFree(lastResult);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "\n[Error] hipFree failed for lastResult!\n");
            goto Error;
        }
        else isLastResultAMalloc = false;
        // �ض����ڴ�
        lastResult = nowResult;
        nowResult = nullptr;
        isNowResultMalloc = false;
        // ����������
        nowDataA = lastResult;
        nowDataB = lastResult + opDataLen;
        // ��ʼ��nowResult�ڴ�
        cudaStatus = hipMalloc((void**)&nowResult, resMemLen * sizeof(float));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "\n[Error] hipMalloc failed for result!\n");
            goto Error;
        }
        else isNowResultMalloc = true;
        // ������׼������
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "\n[Error] device initialization failed: %s!\n", hipGetErrorString(cudaStatus));
            goto Error;
        }

        // ����sum kernel
        sumKernel <<<dimGrid, dimBlock>>> (nowResult, nowDataA, nowDataB);
        // ���kernelִ�д���
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "\n[Error] kernel launching failed: %s @ %d, %d!\n", hipGetErrorString(cudaStatus), dimGrid.x, dimBlock.x);
            goto Error;
        }
        // �ȴ�kernelִ�����
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "\n[Error] hipDeviceSynchronize returned error code %d after launching kernel!\n", cudaStatus);
            goto Error;
        }

        // ��������
        if (isOdd)
        {
            // ���������ڴ�
            cudaStatus = hipMemcpy(nowResult + opDataLen, lastResult + 2 * opDataLen, sizeof(float), hipMemcpyDeviceToDevice);
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "\n[Error] hipMemcpy failed during odd padding!\n");
                goto Error;
            }
        }

        // ѭ������
        wholeDataLen = resDataLen;
    }


    // ����ִ�н��: device -> host
    if (nowResult != nullptr)
    {
        *retLen = wholeDataLen;

        cudaStatus = hipMemcpy(retValue, nowResult, wholeDataLen * sizeof(float), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "\n[Error] hipMemcpy failed when writting back to host!\n");
            goto Error;
        }
    }


    // �ͷ�ָ����ָ�ڴ�!
Error:
    if (isLastResultAMalloc) hipFree(lastResult);
    if (isNowResultMalloc) hipFree(nowResult);

    return cudaStatus;
}


__host__ hipError_t releaseCuda(void)
{
    // ����CUDA�豸, ���̼���!
    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n[Error] hipDeviceReset failed!\n");
    }

    return cudaStatus;
}


__global__ void readKernel(float* retValue, const float* data)
{
    // ��λ
    unsigned int thIndexX = blockIdx.x * blockDim.x + threadIdx.x;
    // ����
    retValue[thIndexX] = log(sqrt(data[thIndexX]));
}


__global__ void sumKernel(float* retValue, const float* dataA, const float* dataB)
{
    // ��λ
    unsigned int thIndexX = blockIdx.x * blockDim.x + threadIdx.x;
    // ����
    retValue[thIndexX] = dataA[thIndexX] + dataB[thIndexX];
}

