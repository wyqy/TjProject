#include "hip/hip_runtime.h"
#include "cuda_sum.cuh"

__host__ hipError_t initialCuda(int device, float* arrRaw, size_t lenRaw, float* arrLoc, size_t lenLoc)
{
    // ��ʼ��CUDA�豸, �̼߳���!
    hipError_t cudaStatus;

    // �����������
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n[Error] last execution failed: %s!\n", hipGetErrorString(cudaStatus));
    }

    // ȷ��CUDA�豸, Ĭ��ֻѡ�е�һ���豸
    cudaStatus = hipSetDevice(device);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n[Error] hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
    }

    // ��ʼ����ҳ�ڴ�
    cudaStatus = hipHostRegister(arrRaw, lenRaw * sizeof(float), hipHostMallocMapped);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n[Error] hipHostRegister for arrRaw failed!\n");
    }
    cudaStatus = hipHostRegister(arrLoc, lenLoc * sizeof(float), hipHostMallocMapped);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n[Error] hipHostRegister for arrLoc failed!\n");
    }

    return cudaStatus;
}


__host__ hipError_t sumWithCuda(float* retValue, size_t* retLen, const float* data, size_t len)
{
    float* lastResult = nullptr;
    float* nowDataA = nullptr;
    float* nowDataB = nullptr;
    float* nowResult = nullptr;
    bool   isLastResultMalloc = false;
    bool   isNowResultMalloc = false;

    bool isOdd = false;
    size_t wholeDataLen = len;
    size_t opDataLen = 0;
    size_t resDataLen = 0;
    size_t resMemLen = 0;

    dim3 dimBlock(BLOCK_DIMONE_x1024);
    dim3 dimGrid(1);

    hipError_t cudaStatus;

    // �����������
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n[Error] last execution failed: %s!\n", hipGetErrorString(cudaStatus));
        goto Error;
    }


    // ���������Դ�
    // ��������size
    resDataLen = wholeDataLen;
    // ����grid��blocks��size
    dimGrid.x = ((unsigned int)resDataLen % BLOCK_DIMONE_x1024 != 0) ? ((unsigned int)resDataLen / BLOCK_DIMONE_x1024 + 1) : ((unsigned int)resDataLen / BLOCK_DIMONE_x1024);
    if ((unsigned int)resDataLen < BLOCK_DIMONE_x1024) dimBlock.x = (unsigned int)resDataLen;
    else dimBlock.x = BLOCK_DIMONE_x1024;
    // �����ڴ�size
    resMemLen = max((size_t)(dimGrid.x * dimBlock.x), resDataLen);


    // ��ʼ����������: host -> device
    // ��ʼ��lastResult�ڴ�, �㿽������!
    cudaStatus = hipHostGetDevicePointer<float>(&lastResult, const_cast<float*>(data), 0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n[Error] hipHostGetDevicePointer failed for source!\n");
        goto Error;
    }


    // ����������
    nowDataA = lastResult;
    nowDataB = lastResult + opDataLen;
    // ��ʼ��nowResult�ڴ�
    cudaStatus = hipMalloc((void**)&nowResult, resMemLen * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n[Error] hipMalloc failed for result!\n");
        goto Error;
    }
    else isNowResultMalloc = true;


    // ����read kernel
    readKernel <<<dimGrid, dimBlock>>> (nowResult, lastResult);
    // ���kernelִ�д���
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n[Error] kernel launching failed: %s @ %d, %d!\n", hipGetErrorString(cudaStatus), dimGrid.x, dimBlock.x);
        goto Error;
    }
    // �ȴ�kernelִ�����
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n[Error] hipDeviceSynchronize returned error code %d after launching kernel!\n", cudaStatus);
        goto Error;
    }


    // ��ʼ���
    // �鲢��: ������С!
    while (wholeDataLen > CUDA_MAX_RESLEN)
    {
        // ��������size
        isOdd = (wholeDataLen % 2 != 0);
        opDataLen = wholeDataLen / 2;
        resDataLen = isOdd ? opDataLen + 1 : opDataLen;
        // ����grid��blocks��size
        dimGrid.x = ((unsigned int)opDataLen % BLOCK_DIMONE_x1024 != 0) ? ((unsigned int)opDataLen / BLOCK_DIMONE_x1024 + 1) : ((unsigned int)opDataLen / BLOCK_DIMONE_x1024);
        if ((unsigned int)opDataLen < BLOCK_DIMONE_x1024) dimBlock.x = (unsigned int)opDataLen;
        else dimBlock.x = BLOCK_DIMONE_x1024;
        // �����ڴ�size
        resMemLen = max((size_t)(dimGrid.x * dimBlock.x), resDataLen);

        // �������ڴ洦��
        // �ͷ�lastResult�ڴ�
        if (isLastResultMalloc) cudaStatus = hipFree(lastResult);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "\n[Error] hipFree failed for lastResult!\n");
            goto Error;
        }
        else isLastResultMalloc = false;
        // �ض����ڴ�
        lastResult = nowResult;
        isLastResultMalloc = true;
        nowResult = nullptr;
        isNowResultMalloc = false;
        // ����������
        nowDataA = lastResult;
        nowDataB = lastResult + opDataLen;
        // ��ʼ��nowResult�ڴ�
        cudaStatus = hipMalloc((void**)&nowResult, resMemLen * sizeof(float));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "\n[Error] hipMalloc failed for result!\n");
            goto Error;
        }
        else isNowResultMalloc = true;
        // ������׼������
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "\n[Error] device initialization failed: %s!\n", hipGetErrorString(cudaStatus));
            goto Error;
        }

        // ����sum kernel
        sumKernel <<<dimGrid, dimBlock>>> (nowResult, nowDataA, nowDataB);
        // ���kernelִ�д���
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "\n[Error] kernel launching failed: %s @ %d, %d!\n", hipGetErrorString(cudaStatus), dimGrid.x, dimBlock.x);
            goto Error;
        }
        // �ȴ�kernelִ�����
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "\n[Error] hipDeviceSynchronize returned error code %d after launching kernel!\n", cudaStatus);
            goto Error;
        }

        // ��������
        if (isOdd)
        {
            // ���������ڴ�
            cudaStatus = hipMemcpy(nowResult + opDataLen, lastResult + 2 * opDataLen, sizeof(float), hipMemcpyDeviceToDevice);
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "\n[Error] hipMemcpy failed during odd padding!\n");
                goto Error;
            }
        }

        // ѭ������
        wholeDataLen = resDataLen;
    }


    // ����ִ�н��: device -> host
    if (nowResult != nullptr)
    {
        *retLen = wholeDataLen;

        cudaStatus = hipMemcpy(retValue, nowResult, wholeDataLen * sizeof(float), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "\n[Error] hipMemcpy failed when writting back to host!\n");
            goto Error;
        }
    }


    // �ͷ�ָ����ָ�ڴ�!
Error:
    if (isLastResultMalloc) hipFree(lastResult);
    if (isNowResultMalloc) hipFree(nowResult);

    return cudaStatus;
}


__host__ hipError_t releaseCuda(void)
{
    // ����CUDA�豸, ���̼���!
    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n[Error] hipDeviceReset failed!\n");
    }

    return cudaStatus;
}


__global__ void readKernel(float* retValue, const float* data)
{
    // ��λ
    unsigned int thIndexX = blockIdx.x * blockDim.x + threadIdx.x;
    // ����
    retValue[thIndexX] = logf(sqrtf(data[thIndexX]));
}


__global__ void sumKernel(float* retValue, const float* dataA, const float* dataB)
{
    // ��λ
    unsigned int thIndexX = blockIdx.x * blockDim.x + threadIdx.x;
    // ����
    retValue[thIndexX] = dataA[thIndexX] + dataB[thIndexX];
}

