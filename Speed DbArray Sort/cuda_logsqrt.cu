#include "hip/hip_runtime.h"
#include "cuda_logsqrt.cuh"

__host__ hipError_t logsqrtWithCuda(float* data, size_t len)
{
    float* gpuMemPtr = nullptr;
    bool   isGpuMemMalloc = false;

    bool isOdd = false;
    size_t opMemLen = 0;

    dim3 dimBlock(BLOCK_DIMONE_x1024);
    dim3 dimGrid(1);

    hipError_t cudaStatus;

    // �����������
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n[Error] last execution failed: %s!\n", hipGetErrorString(cudaStatus));
        goto Error;
    }


    // ���������Դ�
    // ��������size
    // ����grid��blocks��size
    dimGrid.x = ((unsigned int)len % BLOCK_DIMONE_x1024 != 0) ? ((unsigned int)len / BLOCK_DIMONE_x1024 + 1) : ((unsigned int)len / BLOCK_DIMONE_x1024);
    if ((unsigned int)len < BLOCK_DIMONE_x1024) dimBlock.x = (unsigned int)len;
    else dimBlock.x = BLOCK_DIMONE_x1024;
    // �����ڴ�size
    opMemLen = max((size_t)(dimGrid.x * dimBlock.x), len);


    // �����������: host -> device
    // ��ʼ���ڴ�
    cudaStatus = hipMalloc((void**)&gpuMemPtr, opMemLen * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n[Error] hipMalloc failed for source!\n");
        goto Error;
    }
    else isGpuMemMalloc = true;
    // �����ڴ�
    cudaStatus = hipMemcpy(gpuMemPtr, data, len * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n[Error] hipMemcpy failed when copy dataA for the first time!\n");
        goto Error;
    }


    // ����read kernel
    readKernel <<<dimGrid, dimBlock >>> (gpuMemPtr);
    // ���kernelִ�д���
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n[Error] kernel launching failed: %s @ %d, %d!\n", hipGetErrorString(cudaStatus), dimGrid.x, dimBlock.x);
        goto Error;
    }
    // �ȴ�kernelִ�����
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\n[Error] hipDeviceSynchronize returned error code %d after launching kernel!\n", cudaStatus);
        goto Error;
    }


    // ����ִ�н��: device -> host
    if (gpuMemPtr != nullptr)
    {
        cudaStatus = hipMemcpy(data, gpuMemPtr, len * sizeof(float), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "\n[Error] hipMemcpy failed when writting back to host!\n");
            goto Error;
        }
    }


    // �ͷ�ָ����ָ�ڴ�!
Error:
    if (isGpuMemMalloc) hipFree(gpuMemPtr);

    return cudaStatus;
}


__global__ void readKernel(float* data)
{
    // ��λ
    unsigned int thIndexX = blockIdx.x * blockDim.x + threadIdx.x;
    // ����
    data[thIndexX] = log(sqrt(data[thIndexX]));
}

